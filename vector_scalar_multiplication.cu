
#include <hip/hip_runtime.h>
#include<iostream>
#include<vector>

#define ARRAY_SIZE 8
#define SCALAR 10.0
#define BLOCKS 1

__global__ void vectorScalarMultiplication (float * d_array, float scalar)
{
    d_array[threadIdx.x] *= scalar;
}

float h_vector[ARRAY_SIZE];

int main(void)
{
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

   
    for(int i = 0; i < ARRAY_SIZE; i++)
    {
        h_vector[i] = float(i);
    }

    float * d_vector  = NULL;
    
    //allocate memory on GPU
    hipMalloc( (void**) &d_vector, ARRAY_BYTES);

    //transfer data to GPU
    hipMemcpy(d_vector, h_vector, ARRAY_BYTES, hipMemcpyHostToDevice);

    vectorScalarMultiplication <<<BLOCKS, ARRAY_SIZE>>>(d_vector, SCALAR);

    //transfer result from GPU to the application
    hipMemcpy(h_vector, d_vector, ARRAY_BYTES, hipMemcpyDeviceToHost);
    
    //print the result
    for(int i = 0; i < ARRAY_SIZE; i++)
    {
        std::cout << h_vector[i] << std::endl;
    }

    hipFree(d_vector);
    
    hipDeviceReset();

    return 0;
}