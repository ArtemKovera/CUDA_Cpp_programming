#include<filesystem>
#include<iostream>
#include<fstream> 
#include<hip/hip_runtime.h>
#include<unistd.h>

#define FILE_TO_PROCESS "snail.bmp"
#define FILE_RESULT "snail_2.bmp"

#define IMAGE_HEIGHT 256
#define IMAGE_WIDTH  256

#define OFFSET 54

using namespace std;

__global__ void makeBlurryImage(unsigned char * arr, const size_t imageHeight, const size_t imageWidth)
{



    //get index
    int index = threadIdx.x * blockIdx.x;

    //exclude top and bottom pixels from blurring 
    if(index <= imageWidth * 3 && index >= imageWidth * 3 * (imageHeight - 1)) 
    {
        return;
    }  
    
    /*
    //exlude rightmost and leftmost pixels from blurring
    if(index%(imageWidth*3) == 0 || index%(imageWidth*3 + 1) == 0 || index%(imageWidth*3 + 2)== 0 || 
       index%((imageWidth-1)*3) == 0 || index%((imageWidth-1)*3+1) == 0 || index%((imageWidth-1)*3+2) == 0) 
    {
        return;
    }
    */
    
    //printf("index = %d\n", index);
    //read and compute value
    unsigned char temp = (arr[index] + arr[index + 3] + arr[index - 3] + 
                         arr[index - imageWidth * 3] + arr[index - (imageWidth-1) * 3] + arr[index - (imageWidth+1) * 3] +
                         arr[index + imageWidth * 3] + arr[index + (imageWidth-1) * 3] + arr[index + (imageWidth+1) * 3])/9 ;
    
 
    //sync threads before writing
    __syncthreads();
    //printf("index = %d; value = %d\n", index, temp);

    //write computed value
    arr[index] = temp;
    
     
    //for debug
    //printf("index = %d; thredIdx.x = %d; blockIdx.x = %d\n", index, threadIdx.x, blockIdx.x);
}

int main(int argc, char ** argv)
{
    filesystem::path filePath = FILE_TO_PROCESS;

    const unsigned int fileSize = static_cast<unsigned int>(filesystem::file_size(filePath));

    cout << "File size is: " << fileSize << " bytes" << endl;

    ifstream inputFile(filePath, std::ios::binary); 
  
    if (!inputFile.is_open()) 
    { 
        cerr << "Error opening the file!" << endl; 
        return 1; 
    } 
    else
    {
        std::cout << "File has been opened" << std::endl;
    }    

    char * buffer = new char [fileSize];

    inputFile.read(buffer, fileSize);

    if(!inputFile) 
    {
        std::cerr << "Error reading file, could only read " << inputFile.gcount() << " bytes" << std::endl;
    }
    else
    {
        std::cout << "File has been put in buffer" << std::endl;
    } 

    char * h_img = buffer + OFFSET; 



    //For debugging
    //std::cout << "First symbol in ptr is \"" <<  *h_img << "\" " <<std::endl;
    
    char * imgResult = new char [fileSize];

    std::copy(buffer, buffer+OFFSET, imgResult);     
    
    
    //------ CUDA PART ---------------
    char * d_img = nullptr;
    
    hipError_t error = hipMalloc((void**)&d_img, fileSize - OFFSET);

    if(error != hipSuccess)
    {
        cerr << "Failure of GPU memory allocation for d_img" << std::endl;
        hipGetErrorString(error);
        return 1;
    }

    error = hipMemcpy(d_img, h_img, fileSize - OFFSET, hipMemcpyHostToDevice);
    if(error != hipSuccess)
    {
        cerr << "Failure of transforing memory from host to GPU for d_img" << std::endl;
        hipGetErrorString(error);
        return 1;
    }
    
    makeBlurryImage<<<IMAGE_HEIGHT * 3, IMAGE_WIDTH>>>((unsigned char*)d_img, IMAGE_HEIGHT, IMAGE_WIDTH);

    hipError_t err = hipGetLastError();
    if ( err != hipSuccess )
    {
       printf("CUDA Error: %s\n", hipGetErrorString(err));       
    }    

    error = hipMemcpy(imgResult + OFFSET, d_img, fileSize - OFFSET, hipMemcpyDeviceToHost);
    if(error != hipSuccess)
    {
        cerr << "Failure of transforing memory from GPU to host for d_img" << std::endl;
        hipGetErrorString(error);
        return 1;
    }  
    
    
    ofstream result(FILE_RESULT, ofstream::binary);
    
    if (!result.is_open()) 
    { 
        cerr << "Error opening the output file!" << endl; 
        return 1; 
    } 
    else
    {
        std::cout << "Output file has been opened" << std::endl;
    }   


    result.write(imgResult, fileSize);  
    if(!result) 
    {
        std::cerr << "Error writing to output file " << std::endl;
    }
    else
    {
        std::cout << "Output file has been written" << std::endl;
    }     
    
    
    
    hipFree(d_img);    

    
    delete [] buffer;
    delete [] imgResult;

    return 0;
}
